#include "hip/hip_runtime.h"
#include "rwalk.cuh"
#include "helper.cuh"
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <limits>

int32_t* random_walk_dev;
int32_t* node_idx_dev;
float* timestamp_dev;
int32_t* start_idx_dev;

float *cdf_buffer_host;
float *cdf_buffer_dev;

int32_t *node_idx_host_sorted;
float *timestamp_host_sorted;

float *timestamp_dev_sorted;
int32_t *node_idx_dev_sorted;

int32_t *mapping_host;
int32_t *mapping_dev;

float extend_ratio = 0.1;

int threadBlockSize;
hipDeviceProp_t prop;
int count_dev;
bool preprocessing = true;


// assert(err == hipSuccess);

#define cudaCheck(err) { \
	if (err != hipSuccess) { \
		printf("CUDA error: %s: %s, line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
	} \
}

// rand_walk -> [num_of_node, num_of_walk, max_walk_length]
void __global__ singleRandomWalk(int num_of_node, int num_of_walk, int max_walk_length, int32_t* node_idx, float* timestamp, int32_t* start_idx, int32_t* rand_walk, unsigned long long rnumber){
    // assuming grid = 1
    int32_t i =  (blockDim.x * blockIdx.x) + threadIdx.x;
    rnumber = i * (unsigned long long) rnumber + 11;
    if(i >= num_of_node * num_of_walk){
        return;
    }

    int32_t src_node = i / (int32_t) num_of_walk;
    float curr_timestamp = .0f;
    rand_walk[i * max_walk_length + 0] = src_node;

    // printf("start : %lld ; end : %lld; src_node: %lld; num_of_walk : %d; max_walk_length: %d; i : %lld\n", (long long int)start, (long long int)end, (long long int)src_node, num_of_walk, max_walk_length, (long long int)i);
    int32_t start;
    int32_t end;

    int walk_cnt;
    for(walk_cnt = 1; walk_cnt < max_walk_length; walk_cnt ++){
        // ! can be improved
        start = start_idx[src_node];
        end = start_idx[src_node + 1];
        // printf("start: %lld end: %lld\n", (long long int) start, (long long int)end);

        // control divergence
        // range should be [start, end)
        if(start < end){
            float* valid_timestamp = (float*) malloc((end - start) * sizeof(float));
            int32_t* valid_node = (int32_t*) malloc((end - start) * sizeof(int32_t));
            int idx = 0;
            // float cdf[end - start];
            float max_timestamp = timestamp[start];
            float min_timestamp = timestamp[start];
            // ! parallizable
            for(int32_t j = start; j < end; j ++){
                // printf("idx: %lld, timestamp: %f node_idx: %lld\n", (long long int) j, timestamp[j], (long long int) node_idx);
                if(timestamp[j] > curr_timestamp){
                    valid_node[idx] = node_idx[j];
                    valid_timestamp[idx++] = timestamp[j];
                }
                max_timestamp = max(max_timestamp, timestamp[j]);
                min_timestamp = min(min_timestamp, timestamp[j]);
            }
            // printf("idx: %d\n", idx);
            if(!idx){
                free(valid_node);
                free(valid_timestamp);
                break;
            }

            // every timestamp is the same
            // printf("max: %f ; min : %f\n", max_timestamp, min_timestamp);
            if(max_timestamp - min_timestamp >= - 0.0000001 && max_timestamp - min_timestamp <= 0.0000001){
                // printf("valid node: %lld\n", (long long int)valid_node[0]);
                // printf("Time interval too small\n");
                rand_walk[i * max_walk_length + walk_cnt] = valid_node[0];
                src_node = valid_node[0];
                curr_timestamp = valid_timestamp[0];
                free(valid_node);
                free(valid_timestamp);
                continue;
            }

            float* cdf = (float*) malloc(idx * sizeof(float));

            // ! need to determine how to get prob
            float prob = rnumber * 1.0 / ULLONG_MAX;

            // refresh rnumber
            rnumber = rnumber * (unsigned long long)25214903917 + 11;
            bool fall_through = true;

            // ! reduction tree here (kernel in kernel)
            float denom = .0f;
            for(int j = 0; j < idx; j ++){
                cdf[j] =  expf((valid_timestamp[j] - curr_timestamp) / (max_timestamp - min_timestamp));
                denom += cdf[j];
            }
            float curr_cdf = .0f,  next_cdf = .0f;
            for(int j = 0; j < idx; j ++){
                next_cdf += cdf[j] / denom;
                if(prob >= curr_cdf && prob <= next_cdf){
                    // printf("valid node: %lld\n", (long long int)valid_node[j]);
                    rand_walk[i * max_walk_length + walk_cnt] = valid_node[j];
                    src_node = valid_node[j];
                    curr_timestamp = valid_timestamp[j];
                    fall_through = false;
                    break;
                }
                curr_cdf = next_cdf;
            }

            // fall through should never happen
            if(fall_through){
                // printf("valid node: %lld\n", (long long int)valid_node[0]);
                rand_walk[i * max_walk_length + walk_cnt] = valid_node[0];
                src_node = valid_node[0];
                curr_timestamp = valid_timestamp[0];
            }

            free(valid_node);
            free(valid_timestamp);
            free(cdf);
        }
        else{
            break;
        }
    }

    if(walk_cnt < max_walk_length){
        // signal the rest is invalid and there is no descending node
        rand_walk[i * max_walk_length + walk_cnt] = -1;
    }
}


void cuda_rwalk(int max_walk_length, int num_walks_per_node, int32_t num_nodes, int32_t num_edges, unsigned long long random_number){

#if defined(DEBUG)
    size_t free_memory;
    size_t total_memory;

    cudaCheck(hipMemGetInfo(&free_memory, &total_memory));
    // printf("free memory : %zu ; total memory : %zu\n", free_memory, total_memory);
#endif

    // malloc GPU memory
    cudaCheck(hipMalloc((void **)&start_idx_dev, sizeof(int32_t) * (num_nodes + 1)));
    cudaCheck(hipMalloc((void **)&node_idx_dev, sizeof(int32_t) * num_edges));
    cudaCheck(hipMalloc((void **)&timestamp_dev, sizeof(float) * num_edges));
    cudaCheck(hipMalloc((void **)&random_walk_dev, sizeof(int32_t) * num_nodes * max_walk_length * num_walks_per_node));

    // memcpy
    cudaCheck(hipMemcpy(start_idx_dev, start_idx_host, sizeof(int32_t) * (num_nodes + 1), hipMemcpyHostToDevice));

    hipGetDeviceProperties(&prop, 0);
    threadBlockSize = prop.maxThreadsPerBlock;

    if(preprocessing){
        cdf_buffer_host = new float[num_edges];
        node_idx_host_sorted = new int32_t[num_edges];
        timestamp_host_sorted = new float[num_edges];
        mapping_host = new int32_t[num_edges];

        cudaCheck(hipMalloc((void **)&cdf_buffer_dev, sizeof(float) * num_edges));
        cudaCheck(hipMalloc((void **)&mapping_dev, sizeof(int32_t) * num_edges));
        cudaCheck(hipMalloc((void **)&node_idx_dev_sorted, sizeof(int32_t) * num_edges));
        cudaCheck(hipMalloc((void **)&timestamp_dev_sorted, sizeof(float) * num_edges));
        cuda_helper(max_walk_length, num_walks_per_node, num_nodes, num_edges);
    }
    else{
        cudaCheck(hipMemcpy(node_idx_dev, node_idx_host, sizeof(int32_t) * num_edges, hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(timestamp_dev, timestamp_host, sizeof(float) * num_edges, hipMemcpyHostToDevice));
    }

#if defined(DEBUG)
    hipGetDeviceCount(&count_dev);
    for(int i = 0; i < count_dev; i ++){
        printf("total_global_Mem: %zu MB\n shared_mem_per_block: %zu\n max_threads_per_block: %d\n max_thread_dim: [%d, %d, %d]\n max_grid_size: [%d, %d, %d]",
        prop.totalGlobalMem / 1024 / 1024, prop.sharedMemPerBlock, prop.maxThreadsPerBlock, prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2], prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    }
#endif


    // start training
    int grid_size = (num_nodes * num_walks_per_node - 1) / 32 + 1;
    dim3 gridDim(grid_size);
    dim3 blockDim(32);

    singleRandomWalk<<<gridDim, blockDim>>>(num_nodes, num_walks_per_node, max_walk_length, node_idx_dev, timestamp_dev, start_idx_dev, random_walk_dev, random_number);

#if defined(DEBUG)
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }
#endif

    // cudaCheck(hipMemGetInfo(&free_memory, &total_memory));
    // printf("free memory : %zu ; total memory : %zu\n", free_memory, total_memory);

    // get result
    hipDeviceSynchronize();
    cudaCheck(hipMemcpy(random_walk_host, random_walk_dev, sizeof(int32_t) * num_nodes * max_walk_length * num_walks_per_node, hipMemcpyDeviceToHost));

    if(preprocessing){
        cudaCheck(hipFree(cdf_buffer_dev));
        cudaCheck(hipFree(mapping_dev));
        cudaCheck(hipFree(node_idx_dev_sorted));
        cudaCheck(hipFree(timestamp_dev_sorted));
        delete[] mapping_host;
        delete[] timestamp_host_sorted;
        delete[] node_idx_host_sorted;
        delete[] cdf_buffer_host;
    }
    // clean arrays
    cudaCheck(hipFree(start_idx_dev));
    cudaCheck(hipFree(node_idx_dev));
    cudaCheck(hipFree(timestamp_dev));
    cudaCheck(hipFree(random_walk_dev));
}

