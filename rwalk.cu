#include "hip/hip_runtime.h"
#include "rwalk.cuh"
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <limits>

int64_t* dev_global_walk;
int64_t* dev_node_idx;
float* dev_timestamp;
int64_t* dev_start_idx;

int threadBlockSize;
hipDeviceProp_t prop;


// assert(err == hipSuccess);

#define cudaCheck(err) { \
	if (err != hipSuccess) { \
		printf("CUDA error: %s: %s, line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
	} \
}

// rand_walk -> [num_of_node, num_of_walk, max_walk_length]
void __global__ singleRandomWalk(int num_of_node, int num_of_walk, int max_walk_length, int64_t* node_idx, float* timestamp, int64_t* start_idx, int64_t* rand_walk, unsigned long long rnumber){
    // assuming grid = 1
    int64_t i =  (blockDim.x * blockIdx.x) + threadIdx.x;
    rnumber = i * (unsigned long long) rnumber + 11;
    if(i >= num_of_node * num_of_walk){
        return;
    }

    int64_t src_node = i / (int64_t) num_of_walk;
    float curr_timestamp = .0f;
    rand_walk[i * max_walk_length + 0] = src_node;

    // printf("start : %lld ; end : %lld; src_node: %lld; num_of_walk : %d; max_walk_length: %d; i : %lld\n", (long long int)start, (long long int)end, (long long int)src_node, num_of_walk, max_walk_length, (long long int)i);
    int64_t start;
    int64_t end;

    int walk_cnt;
    for(walk_cnt = 1; walk_cnt < max_walk_length; walk_cnt ++){
        // ! can be improved
        start = start_idx[src_node];
        end = start_idx[src_node + 1];
        // printf("start: %lld end: %lld\n", (long long int) start, (long long int)end);

        // control divergence
        // range should be [start, end)
        if(start < end){
            float* valid_timestamp = (float*) malloc((end - start) * sizeof(float));
            int64_t* valid_node = (int64_t*) malloc((end - start) * sizeof(int64_t));
            int idx = 0;
            // float cdf[end - start];
            float max_timestamp = timestamp[start];
            float min_timestamp = timestamp[start];
            // ! parallizable
            for(int64_t j = start; j < end; j ++){
                // printf("idx: %lld, timestamp: %f node_idx: %lld\n", (long long int) j, timestamp[j], (long long int) node_idx);
                if(timestamp[j] > curr_timestamp){
                    valid_node[idx] = node_idx[j];
                    valid_timestamp[idx++] = timestamp[j];
                }
                max_timestamp = max(max_timestamp, timestamp[j]);
                min_timestamp = min(min_timestamp, timestamp[j]);
            }
            // printf("idx: %d\n", idx);
            if(!idx){
                free(valid_node);
                free(valid_timestamp);
                break;
            }

            // every timestamp is the same
            // printf("max: %f ; min : %f\n", max_timestamp, min_timestamp);
            if(max_timestamp - min_timestamp >= - 0.0000001 && max_timestamp - min_timestamp <= 0.0000001){
                // printf("valid node: %lld\n", (long long int)valid_node[0]);
                // printf("Time interval too small\n");
                rand_walk[i * max_walk_length + walk_cnt] = valid_node[0];
                src_node = valid_node[0];
                curr_timestamp = valid_timestamp[0];
                free(valid_node);
                free(valid_timestamp);
                continue;
            }

            float* cdf = (float*) malloc(idx * sizeof(float));

            // ! need to determine how to get prob
            float prob = rnumber * 1.0 / ULLONG_MAX;

            // refresh rnumber
            rnumber = rnumber * (unsigned long long)25214903917 + 11;
            bool fall_through = true;

            // ! reduction tree here (kernel in kernel)
            float denom = .0f;
            for(int j = 0; j < idx; j ++){
                cdf[j] =  expf((valid_timestamp[j] - curr_timestamp) / (max_timestamp - min_timestamp));
                denom += cdf[j];
            }
            float curr_cdf = .0f,  next_cdf = .0f;
            for(int j = 0; j < idx; j ++){
                next_cdf += cdf[j] / denom;
                if(prob >= curr_cdf && prob <= next_cdf){
                    // printf("valid node: %lld\n", (long long int)valid_node[j]);
                    rand_walk[i * max_walk_length + walk_cnt] = valid_node[j];
                    src_node = valid_node[j];
                    curr_timestamp = valid_timestamp[j];
                    fall_through = false;
                    break;
                }
                curr_cdf = next_cdf;
            }

            // fall through should never happen
            if(fall_through){
                // printf("valid node: %lld\n", (long long int)valid_node[0]);
                rand_walk[i * max_walk_length + walk_cnt] = valid_node[0];
                src_node = valid_node[0];
                curr_timestamp = valid_timestamp[0];
            }

            free(valid_node);
            free(valid_timestamp);
            free(cdf);
        }
        else{
            break;
        }
    }

    if(walk_cnt < max_walk_length){
        // signal the rest is invalid and there is no descending node
        rand_walk[i * max_walk_length + walk_cnt] = -1;
    }
}

void __global__ multipleRandomWalk(int num_of_node, int num_of_walk, int max_walk_length, int64_t* node_idx, float* timestamp, int64_t* start_idx, int64_t* rand_walk, unsigned long long rnumber){
    // assuming grid = 1
    int64_t src_node_original =  (blockDim.x * blockIdx.x) + threadIdx.x;
    if(src_node_original >= num_of_node){
        return;
    }
    rnumber = rnumber * (unsigned long long) src_node_original + 11;

    for (int k = 0; k < num_of_walk; k++)
    {
      int i = src_node_original * num_of_walk + k;
      int64_t src_node = src_node_original;

      float curr_timestamp = .0f;
      rand_walk[i * max_walk_length + 0] = src_node;
    //   printf("In node : %lld\n", (unsigned long long)src_node);

      // printf("start : %lld ; end : %lld; src_node: %lld; num_of_walk : %d; max_walk_length: %d; i : %lld\n", (long long int)start, (long long int)end, (long long int)src_node, num_of_walk, max_walk_length, (long long int)i);
      int64_t start;
      int64_t end;

      int walk_cnt;
      for(walk_cnt = 1; walk_cnt < max_walk_length; walk_cnt ++){
          // ! can be improved
          start = start_idx[src_node];
          end = start_idx[src_node + 1];
          // printf("start: %lld end: %lld\n", (long long int) start, (long long int)end);

          // control divergence
          // range should be [start, end)
          if(start < end){
              float* valid_timestamp = (float*) malloc((end - start) * sizeof(float));
              int64_t* valid_node = (int64_t*) malloc((end - start) * sizeof(int64_t));
              int idx = 0;
              // float cdf[end - start];
              float max_timestamp = timestamp[start];
              float min_timestamp = timestamp[start];
              // ! parallizable
              for(int64_t j = start; j < end; j ++){
                  // printf("idx: %lld, timestamp: %f node_idx: %lld\n", (long long int) j, timestamp[j], (long long int) node_idx);
                  if(timestamp[j] > curr_timestamp){
                      valid_node[idx] = node_idx[j];
                      valid_timestamp[idx++] = timestamp[j];
                  }
                  max_timestamp = max(max_timestamp, timestamp[j]);
                  min_timestamp = min(min_timestamp, timestamp[j]);
              }
              // printf("idx: %d\n", idx);
              if(!idx){
                  free(valid_node);
                  free(valid_timestamp);
                  break;
              }

              // every timestamp is the same
              // printf("max: %f ; min : %f\n", max_timestamp, min_timestamp);
              if(max_timestamp - min_timestamp >= - 0.0000001 && max_timestamp - min_timestamp <= 0.0000001){
                  // printf("valid node: %lld\n", (long long int)valid_node[0]);
                  // printf("Time interval too small\n");
                  rand_walk[i * max_walk_length + walk_cnt] = valid_node[0];
                  src_node = valid_node[0];
                  curr_timestamp = valid_timestamp[0];
                  free(valid_node);
                  free(valid_timestamp);
                  continue;
              }

              float* cdf = (float*) malloc(idx * sizeof(float));

              // ! need to determine how to get prob
              float prob = rnumber * 1.0 / ULLONG_MAX;

              // refresh rnumber
              rnumber = rnumber * (unsigned long long)25214903917 + 11;
              bool fall_through = true;

              // ! reduction tree here (kernel in kernel)
              float denom = .0f;
              for(int j = 0; j < idx; j ++){
                  cdf[j] =  expf((valid_timestamp[j] - curr_timestamp) / (max_timestamp - min_timestamp));
                  denom += cdf[j];
              }
              float curr_cdf = .0f,  next_cdf = .0f;
              for(int j = 0; j < idx; j ++){
                  next_cdf += cdf[j] / denom;
                  if(prob >= curr_cdf && prob <= next_cdf){
                      // printf("valid node: %lld\n", (long long int)valid_node[j]);
                      rand_walk[i * max_walk_length + walk_cnt] = valid_node[j];
                      src_node = valid_node[j];
                      curr_timestamp = valid_timestamp[j];
                      fall_through = false;
                      break;
                  }
                  curr_cdf = next_cdf;
              }

              // fall through should never happen
              if(fall_through){
                  // printf("valid node: %lld\n", (long long int)valid_node[0]);
                  rand_walk[i * max_walk_length + walk_cnt] = valid_node[0];
                  src_node = valid_node[0];
                  curr_timestamp = valid_timestamp[0];
              }

              free(valid_node);
              free(valid_timestamp);
              free(cdf);
          }
          else{
              break;
          }
      }

      if(walk_cnt < max_walk_length){
          // signal the rest is invalid and there is no descending node
          rand_walk[i * max_walk_length + walk_cnt] = -1;
      }
    }
  }

  void __global__ multipleRandomWalk2(int num_of_node, int num_of_walk, int max_walk_length, int64_t* node_idx, float* timestamp, int64_t* start_idx, int64_t* rand_walk, unsigned long long rnumber){
    // assuming grid = 1
    int walk =  (blockDim.x * blockIdx.x) + threadIdx.x;
    if(walk >= num_of_walk){
        return;
    }
    rnumber = rnumber * (unsigned long long) walk + 11;

    for (int k = 0; k < num_of_node; k++)
    {
      int i = k * num_of_walk + walk;
      int64_t src_node = k;

      float curr_timestamp = .0f;
      rand_walk[i * max_walk_length + 0] = src_node;
    //   printf("In node : %lld\n", (unsigned long long)src_node);

      // printf("start : %lld ; end : %lld; src_node: %lld; num_of_walk : %d; max_walk_length: %d; i : %lld\n", (long long int)start, (long long int)end, (long long int)src_node, num_of_walk, max_walk_length, (long long int)i);
      int64_t start;
      int64_t end;

      int walk_cnt;
      for(walk_cnt = 1; walk_cnt < max_walk_length; walk_cnt ++){
          // ! can be improved
          start = start_idx[src_node];
          end = start_idx[src_node + 1];
          // printf("start: %lld end: %lld\n", (long long int) start, (long long int)end);

          // control divergence
          // range should be [start, end)
          if(start < end){
              float* valid_timestamp = (float*) malloc((end - start) * sizeof(float));
              int64_t* valid_node = (int64_t*) malloc((end - start) * sizeof(int64_t));
              int idx = 0;
              // float cdf[end - start];
              float max_timestamp = timestamp[start];
              float min_timestamp = timestamp[start];
              // ! parallizable
              for(int64_t j = start; j < end; j ++){
                  // printf("idx: %lld, timestamp: %f node_idx: %lld\n", (long long int) j, timestamp[j], (long long int) node_idx);
                  if(timestamp[j] > curr_timestamp){
                      valid_node[idx] = node_idx[j];
                      valid_timestamp[idx++] = timestamp[j];
                  }
                  max_timestamp = max(max_timestamp, timestamp[j]);
                  min_timestamp = min(min_timestamp, timestamp[j]);
              }
              // printf("idx: %d\n", idx);
              if(!idx){
                  free(valid_node);
                  free(valid_timestamp);
                  break;
              }

              // every timestamp is the same
              // printf("max: %f ; min : %f\n", max_timestamp, min_timestamp);
              if(max_timestamp - min_timestamp >= - 0.0000001 && max_timestamp - min_timestamp <= 0.0000001){
                  // printf("valid node: %lld\n", (long long int)valid_node[0]);
                  // printf("Time interval too small\n");
                  rand_walk[i * max_walk_length + walk_cnt] = valid_node[0];
                  src_node = valid_node[0];
                  curr_timestamp = valid_timestamp[0];
                  free(valid_node);
                  free(valid_timestamp);
                  continue;
              }

              float* cdf = (float*) malloc(idx * sizeof(float));

              // ! need to determine how to get prob
              float prob = rnumber * 1.0 / ULLONG_MAX;

              // refresh rnumber
              rnumber = rnumber * (unsigned long long)25214903917 + 11;
              bool fall_through = true;

              // ! reduction tree here (kernel in kernel)
              float denom = .0f;
              for(int j = 0; j < idx; j ++){
                  cdf[j] =  expf((valid_timestamp[j] - curr_timestamp) / (max_timestamp - min_timestamp));
                  denom += cdf[j];
              }
              float curr_cdf = .0f,  next_cdf = .0f;
              for(int j = 0; j < idx; j ++){
                  next_cdf += cdf[j] / denom;
                  if(prob >= curr_cdf && prob <= next_cdf){
                      // printf("valid node: %lld\n", (long long int)valid_node[j]);
                      rand_walk[i * max_walk_length + walk_cnt] = valid_node[j];
                      src_node = valid_node[j];
                      curr_timestamp = valid_timestamp[j];
                      fall_through = false;
                      break;
                  }
                  curr_cdf = next_cdf;
              }

              // fall through should never happen
              if(fall_through){
                  // printf("valid node: %lld\n", (long long int)valid_node[0]);
                  rand_walk[i * max_walk_length + walk_cnt] = valid_node[0];
                  src_node = valid_node[0];
                  curr_timestamp = valid_timestamp[0];
              }

              free(valid_node);
              free(valid_timestamp);
              free(cdf);
          }
          else{
              break;
          }
      }

      if(walk_cnt < max_walk_length){
          // signal the rest is invalid and there is no descending node
          rand_walk[i * max_walk_length + walk_cnt] = -1;
      }
    }
  }


void cuda_rwalk(int max_walk_length, int num_walks_per_node, int64_t num_nodes, int64_t num_edges, unsigned long long random_number){

    size_t free_memory;
    size_t total_memory;

    cudaCheck(hipMemGetInfo(&free_memory, &total_memory));
    // printf("free memory : %zu ; total memory : %zu\n", free_memory, total_memory);

    // malloc GPU memory
    cudaCheck(hipMalloc((void **)&dev_start_idx, sizeof(int64_t) * (num_nodes + 1)));
    cudaCheck(hipMalloc((void **)&dev_node_idx, sizeof(int64_t) * num_edges));
    cudaCheck(hipMalloc((void **)&dev_timestamp, sizeof(float) * num_edges));
    cudaCheck(hipMalloc((void **)&dev_global_walk, sizeof(int64_t) * num_nodes * max_walk_length * num_walks_per_node));

    // memcpy
    cudaCheck(hipMemcpy(dev_start_idx, start_idx_host, sizeof(int64_t) * (num_nodes + 1), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dev_node_idx, node_idx_host, sizeof(int64_t) * num_edges, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dev_timestamp, timestamp_host, sizeof(float) * num_edges, hipMemcpyHostToDevice));

    // hipGetDeviceCount(&dev_count);
    // for(int i = 0; i < dev_count; i ++){
    //     printf("total_global_Mem: %zu\n shared_mem_per_block: %zu\n max_threads_per_block: %d\n max_thread_dim: %d\n max_grid_size: %d",
    //     prop.totalGlobalMem, prop.sharedMemPerBlock, prop.maxThreadsPerBlock, prop.maxThreadsDim, prop.maxGridSize);

    // }
    hipGetDeviceProperties(&prop, 0);
    threadBlockSize = prop.maxThreadsPerBlock;
    // start training
    // int grid_size = (num_nodes * num_walks_per_node - 1 ) / 32 + 1;
    int grid_size = (num_nodes - 1) / 32 + 1;
    // int grid_size = (num_walks_per_node - 1) / 32 + 1;
    // printf("grid_size: %d\n", grid_size);
    dim3 gridDim(grid_size);
    dim3 blockDim(32);
    // ?? header file
    multipleRandomWalk<<<gridDim, blockDim>>>(num_nodes, num_walks_per_node, max_walk_length, dev_node_idx, dev_timestamp, dev_start_idx, dev_global_walk, random_number);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }

    // cudaCheck(hipMemGetInfo(&free_memory, &total_memory));
    // printf("free memory : %zu ; total memory : %zu\n", free_memory, total_memory);

    // get result
    hipDeviceSynchronize();
    cudaCheck(hipMemcpy(random_walk_host, dev_global_walk, sizeof(int64_t) * num_nodes * max_walk_length * num_walks_per_node, hipMemcpyDeviceToHost));

    // clean arrays
    cudaCheck(hipFree(dev_start_idx));
    cudaCheck(hipFree(dev_node_idx));
    cudaCheck(hipFree(dev_timestamp));
    cudaCheck(hipFree(dev_global_walk));
}

